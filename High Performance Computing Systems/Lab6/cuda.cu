#include "hip/hip_runtime.h"
#include "utility.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "numgen.c"

__device__
int isPrime(unsigned long int number)
{
	if(number <= 1) return 0;
	for(unsigned long int i = 2; i * i <= number; i++)
	{
		if(number % i == 0) return 0;
	}
	return 1;
}

__global__
void checkPrimes(unsigned long int* numbers, int* results, long n)
{
	long idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < n)
	{
		results[idx] = isPrime(numbers[idx]);
	}
}

int main(int argc,char **argv) {

	Args ins__args;
	parseArgs(&ins__args, &argc, argv);
	
	//program input argument
	long inputArgument = ins__args.arg; 
	unsigned long int *numbers = (unsigned long int*)malloc(inputArgument * sizeof(unsigned long int));
	numgen(inputArgument, numbers);

	struct timeval ins__tstart, ins__tstop;
	gettimeofday(&ins__tstart, NULL);
	
	// run your CUDA kernel(s) here

	unsigned long int* device_numbers;
	int* device_results;
	int* host_results = (int*)malloc(inputArgument * sizeof(int));

	hipMalloc((void**) &device_numbers, inputArgument * sizeof(unsigned long int));
	hipMalloc((void**) &device_results, inputArgument * sizeof(int));
	hipMemcpy(device_numbers, numbers, inputArgument * sizeof(unsigned long int), hipMemcpyHostToDevice);

	int threads = 256;
	int blocks = (inputArgument + threads - 1) / threads;
	checkPrimes<<<blocks, threads>>>(device_numbers, device_results, inputArgument);

	hipMemcpy(host_results, device_results, inputArgument * sizeof(int), hipMemcpyDeviceToHost);

	long result = 0;
	for(long i = 0; i<inputArgument; i++)
	{
		result += host_results[i];
	}

	printf("result %ld", result);

	hipFree(device_numbers);
	hipFree(device_results);
	free (host_results);

	// synchronize/finalize your CUDA computations

	gettimeofday(&ins__tstop, NULL);
	ins__printtime(&ins__tstart, &ins__tstop, ins__args.marker);


}
